#include <stdio.h>
#include <stdint.h>
#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include "support.h"
#include "lzkernel.cu"
#define BLOCK_SIZE 512

using namespace std;

int main(int argc, char** argv) {

    int s_b_s;
    int u_b_s;

    s_b_s = 8;
    u_b_s = 6;

    if(argc != 2) {
	cout << "Invalid number of arguments. Usage is: ./compressFile <filename>\n";
	return 0;
    }
    string fileName = argv[1];
    ifstream inFile(fileName.c_str());

    //Get the length of the file in characters
    unsigned int num_elements = 0;
    if(inFile) {
	inFile.seekg(0, inFile.end);
	num_elements = inFile.tellg();
	inFile.seekg(0, inFile.beg);
    }

    //Read file in character array
    char* h_input = new char[num_elements];
    inFile.read(h_input, num_elements);
    inFile.close();

    char* h_output = new char[2 * 499];

    printf("Allocating device variables...\n");
    char* d_input;
    char* d_output;

 //   unsigned int* d_bins;
    hipError_t cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(char));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for character buffer\n");
    hipMalloc((void**)&d_output,2 * 499 * sizeof(unsigned int));
  // if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for d_bins\n");

    printf("Copying host variables to device...\n");
    cuda_ret = hipMemcpy(d_input, h_input, num_elements * sizeof(char), hipMemcpyHostToDevice);
    //printCharArray(h_input, num_elements);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device\n");
    cuda_ret = hipMemset(d_output, 0, 2 * 499 * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory\n");

    hipDeviceSynchronize();

    // Launch kernel ----------------------------------------------------------
   printf("Launching kernel..."); fflush(stdout);
  //  startTime(&timer);

    dim3 dimGrid((num_elements/BLOCK_SIZE),1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    lz77kernel<<<dimGrid, dimBlock>>>(d_input, d_output, s_b_s, u_b_s); 
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel\n");

    printf("Copying device variables to host...\n");
    cuda_ret = hipMemcpy(h_output, d_output,2 * 499 * sizeof(int), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host\n");
    hipDeviceSynchronize();
 int n ;
for ( n=0; n < (2 * 499 ); ++n)
    cout << h_output[n] << ' ';
    cout << n;
    cout << '\n';
   int count = 0;
 	int r ;
for ( r = 0; r < (2* 499); r = r + 2)
{
//	count++;
//	printf("count = %c\n",h_output[r]);
	if ( h_output[r] != '1')
	{
		int x = h_output[ r + 1] - '0';
		r = 2*(x-1) + r;
	}
	count++;
}
printf("count = %d\n",count);
 
   printf("Freeing memory...\n");
    delete[] h_input, h_output;
    hipFree(d_input);
    hipFree(d_output);
}
