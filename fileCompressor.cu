#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include "support.h"
#include "kernel.cu"
#include "huffmanNode.h"
#include "huffmanUtil.h"

//extern const int NUM_BINS;

using namespace std;

int main(int argc, char** argv) {
    if(argc != 2) {
	cout << "Invalid number of arguments. Usage is: ./compressFile <filename>\n";
	return 0;
    }
    string fileName = argv[1];
    ifstream inFile(fileName.c_str());

    //Get the length of the file in characters
    unsigned int num_elements = 0;
    if(inFile) {
	inFile.seekg(0, inFile.end);
	num_elements = inFile.tellg();
	inFile.seekg(0, inFile.beg);
    }

    //Read file in character array
    char* h_input = new char[num_elements];
    inFile.read(h_input, num_elements);
    inFile.close();

    unsigned int* h_bins = new unsigned int[NUM_BINS];

    printf("Allocating device variables...\n");
    char* d_input;
    unsigned int* d_bins;
    hipError_t cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(char));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for character buffer\n");
    cuda_ret = hipMalloc((void**)&d_bins, NUM_BINS * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for d_bins\n");

    printf("Copying host variables to device...\n");
    cuda_ret = hipMemcpy(d_input, h_input, num_elements * sizeof(char), hipMemcpyHostToDevice);
    //printCharArray(h_input, num_elements);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device\n");
    cuda_ret = hipMemset(d_bins, 0, NUM_BINS * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory\n");

    hipDeviceSynchronize();

    printf("Launching kernel...\n");
    histogram(d_input, d_bins, num_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel\n");

    printf("Copying device variables to host...\n");
    cuda_ret = hipMemcpy(h_bins, d_bins, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host\n");
    hipDeviceSynchronize();
   
    /*for(int i=0; i<NUM_BINS; i++) {
	if(i>=0 && i <= 25) printf("%c frequency is %d \n", i + 97, h_bins[i]);
	else if(i>= 26 && i <= 35) printf("%c frequency is %d \n", i + 22, h_bins[i]);
	else if(i == 36) printf("\" \" frequency is %d \n", h_bins[i]);
	else if(i == 37) printf(". frequency is %d \n", h_bins[i]);
 	else if(i == 38) printf(", frequency is %d \n", h_bins[i]);
	else if(i == 39) printf("\\n frequency is %d \n", h_bins[i]);
    }*/

    //Build encoding tree
    HuffmanNode* encodingTree = buildEncodingTree(h_bins);

    //Build encoding array, store their lengths in h_codeLengths array and initialize it to zero.
    int* h_codeLengths = new int[NUM_BINS]; 
    for(int i = 0; i < NUM_BINS; i++) {
	h_codeLengths[i] = 0;
    }

    string* codeArr = buildEncodingArray(encodingTree, h_codeLengths);

    /*for(int i = 0; i < NUM_BINS; i++) {
	cout << "Code for " << i << ": " << h_codeLengths[i] << endl; 
    }*/

    //Find maximum length of codes
    int maxLength = getMaxCodeLength(codeArr);

    //Convert string* to char**
    char** h_characterCodes = new char*[NUM_BINS];
    for(int i = 0; i < NUM_BINS; i++) {
        char* code = (char*) codeArr[i].c_str();
	h_characterCodes[i] = code;
	//cout << h_characterCodes[i] << endl;
    }

    hipFree(d_input);
    printf("Copying input and character codes array to device...\n");    
    
    cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(char));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for input\n");
    cuda_ret = hipMemcpy(d_input, h_input, num_elements * sizeof(char), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device\n");
    hipDeviceSynchronize();
    
    char** d_characterCodes;
    cuda_ret = hipMalloc((void***)&d_characterCodes, NUM_BINS * sizeof(char*));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for character codes array\n");
    cuda_ret = hipMemcpy(d_characterCodes, h_characterCodes, NUM_BINS * sizeof(char*), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device\n");
    hipDeviceSynchronize();

    int* d_codeLengths;
    cuda_ret = hipMalloc((void**)&d_codeLengths, NUM_BINS * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for array of code lengths\n");
    cuda_ret = hipMemcpy(d_codeLengths, h_codeLengths, NUM_BINS * sizeof(int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device\n");
    hipDeviceSynchronize();

    char *h_output, *d_output;
    h_output = new char[num_elements * maxLength];
    cuda_ret = hipMalloc((void**)&d_output, num_elements * maxLength * sizeof(char));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory for character output array\n");

    hipDeviceSynchronize();

    printf("Launching the device kernel for enocding input character array..");
    encodeData(d_input, num_elements, d_characterCodes, d_codeLengths, maxLength, d_output);

    hipDeviceSynchronize();
    
    printf("Copying character output array from device to host...");
    cuda_ret = hipMemcpy(h_output, d_output, num_elements * maxLength * sizeof(char), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy output character array from device back to host");

    hipDeviceSynchronize();

    ofstream outfile("example2Out.txt");
    outfile.write(h_output, num_elements*maxLength);
    outfile.close();

    printf("Freeing memory...\n");
    delete[] h_input, h_bins;
    hipFree(d_input);
    hipFree(d_bins);
}

